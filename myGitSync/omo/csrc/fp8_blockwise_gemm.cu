#include <ATen/cuda/HIPContext.h>
#include <cudaTypedefs.h>
#include <cutlass/arch/arch.h>
#include <cutlass/arch/memory.h>
#include <cutlass/arch/mma.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/epilogue/thread/activation.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/threadblock/default_thread_map_tensor_op.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/gemm/gemm.h>
#include <cutlass/gemm/kernel/default_gemm_universal_with_visitor.h>
#include <cutlass/gemm/thread/mma.h>
#include <cutlass/layout/matrix.h>
#include <cutlass/matrix_coord.h>
#include <cutlass/numeric_types.h>
#include <cutlass/tensor_ref.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/tensor_view_io.h>
#include <torch/all.h>

#include <cute/tensor.hpp>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/epilogue/collective/default_epilogue.hpp>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>
#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

#include "utils.h"

using namespace cute;

template <typename SchedulerType, typename OutType, typename TileShape, typename ClusterShape>
void launch_sm90_fp8_blockwise_scaled_mm(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  using ElementAccumulator = float;
  using ElementCompute = float;
  using ElementBlockScale = float;

  using ElementA = cutlass::float_e4m3_t;
  using LayoutA = cutlass::layout::RowMajor;
  constexpr int AlignmentA = 128 / cutlass::sizeof_bits<ElementA>::value;

  using ElementB = cutlass::float_e4m3_t;
  using LayoutB = cutlass::layout::ColumnMajor;
  constexpr int AlignmentB = 128 / cutlass::sizeof_bits<ElementB>::value;

  using ElementC = void;
  using LayoutC = cutlass::layout::RowMajor;
  constexpr int AlignmentC = 128 / cutlass::sizeof_bits<OutType>::value;

  using ElementD = OutType;
  using LayoutD = cutlass::layout::RowMajor;
  constexpr int AlignmentD = AlignmentC;

  using ScaleTileShape = Shape<_1, _128, _128>;
  using ScaleConfig = decltype(cutlass::detail::sm90_trivial_blockwise_scale_config(ScaleTileShape{}));
  using LayoutSFA = decltype(ScaleConfig::deduce_layoutSFA());
  using LayoutSFB = decltype(ScaleConfig::deduce_layoutSFB());

  using ArchTag = cutlass::arch::Sm90;
  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using EpilogueSchedule = cutlass::epilogue::TmaWarpSpecializedCooperative;
  using EpilogueTileType = cutlass::epilogue::collective::EpilogueTileAuto;
  using StoreEpilogueCompute = typename cutlass::epilogue::fusion::Sm90EVT<cutlass::epilogue::fusion::Sm90AccFetch>;

  using KernelSchedule = cutlass::gemm::KernelTmaWarpSpecializedCooperativeFP8BlockScaledAccum;
  using CollectiveEpilogue = typename cutlass::epilogue::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      TileShape,
      ClusterShape,
      EpilogueTileType,
      ElementAccumulator,
      ElementCompute,
      ElementC,
      LayoutC,
      AlignmentC,
      ElementD,
      LayoutD,
      AlignmentD,
      EpilogueSchedule,
      StoreEpilogueCompute>::CollectiveOp;

  using CollectiveMainloop = typename cutlass::gemm::collective::CollectiveBuilder<
      ArchTag,
      OperatorClass,
      ElementA,
      cute::tuple<LayoutA, LayoutSFA>,
      AlignmentA,
      ElementB,
      cute::tuple<LayoutB, LayoutSFB>,
      AlignmentB,
      ElementAccumulator,
      TileShape,
      ClusterShape,
      cutlass::gemm::collective::StageCountAutoCarveout<static_cast<int>(
          sizeof(typename CollectiveEpilogue::SharedStorage))>,
      KernelSchedule>::CollectiveOp;

  using GemmKernel = cutlass::gemm::kernel::GemmUniversal<
      Shape<int, int, int, int>,  // Indicates ProblemShape
      CollectiveMainloop,
      CollectiveEpilogue,
      SchedulerType>;
  using Gemm = cutlass::gemm::device::GemmUniversalAdapter<GemmKernel>;

  Gemm gemm_op;

  int m = a.size(0);
  int k = a.size(1);
  int n = b.size(1);

  auto a_ptr = static_cast<ElementA*>(a.data_ptr());
  auto b_ptr = static_cast<ElementB*>(b.data_ptr());
  auto o_ptr = static_cast<ElementD*>(out.data_ptr());

  auto a_s_ptr = static_cast<ElementBlockScale*>(scales_a.data_ptr());
  auto b_s_ptr = static_cast<ElementBlockScale*>(scales_b.data_ptr());

  using StrideA = typename Gemm::GemmKernel::StrideA;
  using StrideB = typename Gemm::GemmKernel::StrideB;
  using StrideC = typename Gemm::GemmKernel::StrideC;
  using StrideD = typename Gemm::GemmKernel::StrideD;

  StrideA stride_a = cutlass::make_cute_packed_stride(StrideA{}, cute::make_shape(m, k, 1));
  StrideB stride_b = cutlass::make_cute_packed_stride(StrideB{}, cute::make_shape(n, k, 1));
  StrideC stride_c;
  StrideD stride_d = cutlass::make_cute_packed_stride(StrideD{}, cute::make_shape(m, n, 1));

  LayoutSFA layout_sfa = ScaleConfig::tile_atom_to_shape_SFA(make_shape(m, n, k, 1));
  LayoutSFB layout_sfb = ScaleConfig::tile_atom_to_shape_SFB(make_shape(m, n, k, 1));

  typename GemmKernel::MainloopArguments mainloop_args{
      a_ptr, stride_a, b_ptr, stride_b, a_s_ptr, layout_sfa, b_s_ptr, layout_sfb};
  typename GemmKernel::EpilogueArguments epilogue_args{{}, nullptr, stride_d, o_ptr, stride_d};

  typename Gemm::Arguments args = {
      cutlass::gemm::GemmUniversalMode::kGemm,
      {m, n, k, 1},
      mainloop_args,
      epilogue_args,
  };

  size_t workspace_size = gemm_op.get_workspace_size(args);
  auto const workspace_options = torch::TensorOptions().dtype(torch::kUInt8).device(a.device());
  auto workspace = torch::empty(workspace_size, workspace_options);
  auto stream = at::cuda::getCurrentCUDAStream(a.get_device());

  auto can_implement = gemm_op.can_implement(args);
  TORCH_CHECK(can_implement == cutlass::Status::kSuccess, cutlassGetStatusString(can_implement))

  auto status = gemm_op.run(args, workspace.data_ptr(), stream);
  TORCH_CHECK(status == cutlass::Status::kSuccess, cutlassGetStatusString(status))
}



template <typename OutType>
void sm90_fp8_blockwise_dispatch_shape(
    torch::Tensor& out,
    const torch::Tensor& a,
    const torch::Tensor& b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b) {
  using TileShape = Shape<_128, _128, _128>;
  using ClusterShape = Shape<_1, _2, _1>;

  auto k = a.size(1);
  auto n = b.size(1);
  if (k > 3 * n) {
    launch_sm90_fp8_blockwise_scaled_mm<cutlass::gemm::StreamKScheduler, OutType, TileShape, ClusterShape>(
        out, a, b, scales_a, scales_b);
  } else {
    launch_sm90_fp8_blockwise_scaled_mm<cutlass::gemm::PersistentScheduler, OutType, TileShape, ClusterShape>(
        out, a, b, scales_a, scales_b);
  }
}


namespace omo{
torch::Tensor fp8_blockwise_scaled_mm(
    const torch::Tensor& mat_a,
    const torch::Tensor& mat_b,
    const torch::Tensor& scales_a,
    const torch::Tensor& scales_b,
    const torch::Dtype& out_dtype) {
  TORCH_CHECK(mat_a.is_cuda(), "mat_a must be a CUDA tensor");
  TORCH_CHECK(mat_b.is_cuda(), "mat_b must be a CUDA tensor");
  TORCH_CHECK(mat_a.dim() == 2, "mat_a must be a 2D tensor");
  TORCH_CHECK(mat_b.dim() == 2, "mat_b must be a 2D tensor");
  TORCH_CHECK(mat_a.stride(1) == 1, "mat_a must be a row major tensor");
  // TORCH_CHECK(mat_b.stride(0) == 1, "mat_b must be a column major tensor");
  printf("mata = %d,%d\n", mat_a.size(0),mat_a.size(1));
  printf("matb = %d,%d\n", mat_b.size(0),mat_b.size(1));
  printf("sa = %d,%d\n", scales_a.size(0),scales_a.size(1));
  printf("sb = %d,%d\n", scales_b.size(0),scales_b.size(1));
  TORCH_CHECK(mat_a.size(1) == mat_b.size(0), "mat_a and mat_b shapes cannot be multiplied");

  TORCH_CHECK(
      (mat_a.size(1) * mat_a.element_size()) % 16 == 0, "mat_a must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(
      (mat_b.size(0) * mat_b.element_size()) % 16 == 0, "mat_b must be multiple of 16 bytes for memory alignment");
  TORCH_CHECK(mat_a.scalar_type() == torch::kFloat8_e4m3fn, "mat_a must be Float8_e4m3fn");
  TORCH_CHECK(mat_b.scalar_type() == torch::kFloat8_e4m3fn, "mat_b must be Float8_e4m3fn");
  TORCH_CHECK(out_dtype == torch::kHalf || out_dtype == torch::kBFloat16, "out_dtype must be Half or BFloat16");

  auto is_contiguous_vector = [](const torch::Tensor& t) {
    auto t_sizes = t.sizes();
    return t.is_contiguous() &&
           (t.dim() == 1 || (t.dim() == 2 && *std::min_element(t_sizes.begin(), t_sizes.end()) == 1));
  };

  TORCH_CHECK(mat_a.size(0) == scales_a.size(0), "size of scales_a is not matched");
  TORCH_CHECK(mat_a.size(1) / 128 == scales_a.size(1), "size of scales_a is not matched");
  TORCH_CHECK(scales_a.stride(0) == 1 || is_contiguous_vector(scales_a), "scales_a must be M major");
  TORCH_CHECK(mat_b.size(0) / 128 == scales_b.size(0), "size of scales_b is not matched");
  TORCH_CHECK(mat_b.size(1) / 128 == scales_b.size(1), "size of scales_b is not matched");
  TORCH_CHECK(scales_b.stride(0) == 1 || is_contiguous_vector(scales_b), "scales_b must be K major");
  TORCH_CHECK(scales_a.scalar_type() == torch::kFloat32, "scales_a must be Float32");
  TORCH_CHECK(scales_b.scalar_type() == torch::kFloat32, "scales_b must be Float32");

  torch::Tensor out = torch::empty({mat_a.size(0), mat_b.size(1)}, mat_a.options().dtype(out_dtype));
  // torch::Tensor out = torch::empty({mat_a.size(0), mat_b.size(0)}, mat_a.options().dtype(out_dtype));
  TORCH_CHECK((out.size(1) * out.element_size()) % 16 == 0, "out must be multiple of 16 bytes for memory alignment");

  auto sm_version = getSMVersion();

  int64_t original_rows = mat_a.size(0);
  torch::Tensor mat_a_padded = pad_tensor(mat_a, /*alignment=*/4);
  torch::Tensor scales_a_padded = pad_tensor(scales_a, /*alignment=*/4, /*col_major=*/true);
  // torch::Tensor out_padded = torch::empty({mat_a_padded.size(0), mat_b.size(1)}, out.options());
  torch::Tensor out_padded = torch::empty({mat_a_padded.size(0), mat_b.size(0)}, out.options());

#if defined(CUTLASS_ARCH_MMA_SM90_SUPPORTED)
#if defined CUDA_VERSION && CUDA_VERSION >= 12000
  if (sm_version == 90) {
    torch::Tensor scales_b_contiguous = scales_b.contiguous();
    if (out_dtype == torch::kBFloat16) {
      sm90_fp8_blockwise_dispatch_shape<cutlass::bfloat16_t>(
          out_padded, mat_a_padded, mat_b, scales_a_padded, scales_b_contiguous);
    } else {
      sm90_fp8_blockwise_dispatch_shape<cutlass::half_t>(
          out_padded, mat_a_padded, mat_b, scales_a_padded, scales_b_contiguous);
    }
    return out_padded.slice(0, 0, original_rows);
  }
#endif
#endif


  TORCH_CHECK_NOT_IMPLEMENTED(
      false, "No implemented fp8_blockwise_scaled_mm for current compute capability: ", sm_version);
}
}